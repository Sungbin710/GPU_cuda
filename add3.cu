
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void add( int *a, int *b, int *c ) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}


int main( void ) {
	int *a, *b, *c; // host copies of a, b, c
	int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
	int size = N * sizeof( int ); // we need space for an integer

	clock_t start, end;
	double result;

	// allocate device copies of a, b, c
	hipMalloc( (void**)&dev_a, size );
	hipMalloc( (void**)&dev_b, size );
	hipMalloc( (void**)&dev_c, size );

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for(int i=0;i<N;i++)
	{
		a[i] = rand()%100;
		b[i] = rand()%100;
	}

//	random_ints( a, N);
//	random_ints( b, N);

	// copy inputs to device
	hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

	start = clock();
	// launch add() kernel with N parallel blocks 
	add<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( dev_a, dev_b, dev_c );
	end = clock();

	// copy device result back to host copy of c
	hipMemcpy( c, dev_c, size, hipMemcpyDeviceToHost );
	
	free(a); free(b); free(c);
	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );
/*
	for(int i=0; i<N; i++)
		printf("[%d] : %d + %d = %d\n",i,a[i],b[i],c[i]);
*/

	result = (double)(end - start)/CLOCKS_PER_SEC;	
	printf("processing time: %lfs\n",result);

	return 0;
}
